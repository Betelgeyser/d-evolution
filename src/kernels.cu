
#include <hip/hip_runtime.h>
/**
 * Copyright © 2018 Sergei Iurevich Filippov, All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void kernel_tanh(float *x, int n)
{
	for (int i = 0; i < n; i++)
		x[i] = tanhf(x[i]);
}

__host__
void cuda_tanh(float *x, int n)
{
	kernel_tanh<<<1, 1>>>(x, n);
}

